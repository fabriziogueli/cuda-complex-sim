
#include <hip/hip_runtime.h>
/* Copyright (C) 2012  Fabrizio Gueli
 *
 * This file is part of Cuda-complex-sim
 *
 * Cuda-complex-sim is free software: you can redistribute it and/or
 * modify it under the terms of the GNU Lesser General Public
 * License as published by the Free Software Foundation, either
 * version 3 of the License, or (at your option) any later version.
 *
 * Cuda-complex-sim is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
 * Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General Public
 * License along with Cuda-complex-sim.  If not, see <http://www.gnu.org/licenses/>.
 */

/*
#include "device.cuh"
#include "host.hpp"
#include "graph_transf.hpp"
#include "h_barabasi_game.hpp"
#include "hygra.cuh"


int main(int argc, char** argv)
{

		bool* nodes_dev;
		Link* links_target_dev;
		bool* flagw_array;
		bool* flagr_array;
		uint32_t* counter;
		task_t* task_dev;
		task_arguments* task_args_dev;
		message_t* inbox_dev;
		n_attribute *prog;
		curandState *d_state;

//		if (argc!=3)
	//		{
//				perror("\nErrore");
//				exit(1);
//			}


//	uint32_t max_nodes=atoi(argv[1]);
//	uint8_t average_links=atoi(argv[2]);
	uint16_t supplementary_size= 10;




	uint32_t max_nodes = 1000000;
	uint8_t average_links= 2;
	uint16_t barabasi_initial_nodes=average_links+1;

	allocateDataStructures(&prog,&nodes_dev, &task_dev, &task_args_dev, &links_target_dev, &inbox_dev,max_nodes,average_links,supplementary_size,&d_state,&flagw_array,&flagr_array,&counter);
	h_allocateDataStructures(supplementary_size,max_nodes,average_links);

	Graph g = h_barabasi_game(barabasi_initial_nodes, 1, max_nodes);
	startSimulation(links_target_dev,nodes_dev,supplementary_size,g);




	srand(time(NULL));
//	Link init;
//	init.target=-1;
//	init_data<<<BLOCKS,THREADS_PER_BLOCK>>>();
//	h_initArray<bool>(false,h_nodes_array,h_max_nodes_number);
//	h_initArray<Link>(init, h_links_target_array, h_max_nodes_number*h_average_links_number);

//	adjlistToCompactList(g);

//	copyToDevice(nodes_dev,h_nodes_array , 0, h_max_nodes_number );
//	copyToDevice(links_target_dev,h_links_target_array ,0, h_max_nodes_number*h_average_links_number );

  	size_t avail;
  	size_t total;
  	cudaMemGetInfo( &avail, &total );

  	size_t used = total - avail;
  	printf("\nMemoria: totale %d, in uso %d, disponibile: %d", total, used, avail);

  	cudaEvent_t start, stop;
  	cudaEventCreate(&start);
  	cudaEventCreate(&stop);
  	// Start record
  	cudaEventRecord(start, 0);

  		message_test<<<BLOCKS,THREADS_PER_BLOCK,h_average_links_number*THREADS_PER_BLOCK*sizeof(Link)>>>();
  		message_test2nd<<<BLOCKS,THREADS_PER_BLOCK,h_average_links_number*THREADS_PER_BLOCK*sizeof(Link)>>>();
  		message_test2nd<<<BLOCKS,THREADS_PER_BLOCK,h_average_links_number*THREADS_PER_BLOCK*sizeof(Link)>>>();
  		cudaEventRecord(stop, 0);
  		cudaEventSynchronize(stop);
  		float elapsedTime;
  		cudaEventElapsedTime(&elapsedTime, start, stop); // that's our time!
  		// Clean up:
  		cudaEventDestroy(start);
  		cudaEventDestroy(stop);
  		FILE *file;
  		file=fopen("times.txt","a");
  		fprintf(file, "%f\n",elapsedTime);
  		fflush(file);
  		fclose(file);
  		cudaThreadExit();


  return 0;
}

*/
